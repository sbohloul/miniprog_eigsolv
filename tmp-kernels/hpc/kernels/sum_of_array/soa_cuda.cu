#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <numeric>
#include <timer.hpp>

__global__ void sumVector(const double *v_in, double *v_out, int nelem)
{
    int thread_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_idx < nelem)
    {
        v_out[thread_idx] = v_in[thread_idx] + 1.0;
    }
}

constexpr int nelem = 10000000;

int main()
{

    std::vector<double> h_v(nelem, 1.0);
    std::vector<double> h_w(nelem, 0.0);

    // allocate device memory
    double *d_v;
    double *d_w;
    hipMalloc((void **)&d_v, nelem * sizeof(double));
    hipMalloc((void **)&d_w, nelem * sizeof(double));

    // copy to device
    hipMemcpy(d_v, h_v.data(), nelem * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w.data(), nelem * sizeof(double), hipMemcpyHostToDevice);

    // call kernel on device
    int nThreadsPerBlock = 256;
    int nBlocks = (nelem + nThreadsPerBlock - 1) / nThreadsPerBlock;

    sumVector<<<nBlocks, nThreadsPerBlock>>>(d_v, d_w, nelem);
    hipDeviceSynchronize();

    // copy to host
    hipMemcpy(h_w.data(), d_w, nelem * sizeof(double), hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_v);
    hipFree(d_w);

    // check
    double sum = std::accumulate(h_w.begin(), h_w.end(), 0.0);
    std::cout << "h_w[i] = " << sum / nelem << std::endl;

    return 0;
}