#include <iostream>

using namespace std;

int main(int argc, char **argv)
{

    cout << "==========================" << endl;
    cout << "Getting GPU(s) information" << endl;
    cout << "==========================" << endl;

    int driverVersion = 0;
    hipDriverGetVersion(&driverVersion);
    cout << "driverVersion = " << driverVersion << endl;

    int runtimeVersion;
    hipRuntimeGetVersion(&runtimeVersion);
    cout << "runtimeVersion = " << runtimeVersion << endl;

    int numDevices = 0;
    hipError_t cudaStatus = hipGetDeviceCount(&numDevices);

    if (cudaStatus != hipSuccess)
    {
        cerr << "hipGetDeviceCount failed: " << hipGetErrorString(cudaStatus) << endl;
    }
    cout << "numDevices: " << numDevices << endl;
    cout << endl;

    for (int i = 0; i < numDevices; i++)
    {
        hipDeviceProp_t prop;
        cudaStatus = hipGetDeviceProperties(&prop, i);

        cout << "Device " << i << ":" << endl;
        cout << "prop.name " << prop.name << endl;
        cout << "prop.major " << prop.major << endl;
        cout << "prop.minor " << prop.minor << endl;
        cout << "prop.multiProcessorCount " << prop.multiProcessorCount << endl;
        cout << "prop.maxThreadsPerMultiProcessor " << prop.maxThreadsPerMultiProcessor << endl;
        cout << "prop.clockRate  " << prop.clockRate << endl;
        cout << "prop.totalGlobalMem " << prop.totalGlobalMem << endl;
        cout << endl;
    }

    return 0;
}